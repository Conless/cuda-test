
#include <hip/hip_runtime.h>
#include <device_atomic_functions.h>
#include <math.h>

__global__ void dotProduct(int *a, int *b, int *c, int n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n) {
    atomicAdd(c, a[tid] * b[tid]);
  }
}

int main() {
  int n = 100000;
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = n * sizeof(int);

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, sizeof(int));

  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(sizeof(int));

  for (int i = 0; i < n; i++) {
    a[i] = i;
    b[i] = i;
  }

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, sizeof(int), hipMemcpyHostToDevice);

  dotProduct<<<ceil(n / 256.0), 256>>>(d_a, d_b, d_c, n);

  hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost);


  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  free(a);
  free(b);
  free(c);

  return 0;
}
