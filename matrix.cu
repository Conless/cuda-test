
#include <hip/hip_runtime.h>
__global__ void matrixMulKernel(int *d_M, int *d_N, int *d_P, int Width) {
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  if ((Row < Width) && (Col < Width)) {
    int Pvalue = 0;
    for (int k = 0; k < Width; ++k) {
      Pvalue += d_M[Row * Width + k] * d_N[k * Width + Col];
    }
    d_P[Row * Width + Col] = Pvalue;
    
  }
}

int main(int argc, char **argv) {
  int Width = 5;
  int *h_M, *h_N, *h_P;
  int *d_M, *d_N, *d_P;
  int size = Width * Width * sizeof(int);
  h_M = (int *)malloc(size);
  h_N = (int *)malloc(size);
  h_P = (int *)malloc(size);
  hipMalloc((void **)&d_M, size);
  hipMalloc((void **)&d_N, size);
  hipMalloc((void **)&d_P, size);
  for (int i = 0; i < Width * Width; i++) {
    h_M[i] = 1;
    h_N[i] = 2;
  }
  hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
  hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
  dim3 dimGrid(1, 1);
  dim3 dimBlock(Width, Width);
  matrixMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, Width);
  hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
  hipFree(d_M);
  hipFree(d_N);
  hipFree(d_P);
  free(h_M);
  free(h_N);
  free(h_P);
  return 0;
}